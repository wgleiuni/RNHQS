#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <complex>
#include <string>
#include <fstream>
#include <time.h>
#include <hip/hip_runtime.h>


#define Nblock 16
#define NThreadPerBlock 64
#define NTotal Nblock*NThreadPerBlock
#define N_ 200
class RNHQS_ZB
{
    public:
        __device__ RNHQS_ZB();
        __device__ void initial(int,double,double,double,double,double,int,double*,double*,double*,double*,double*,double*,double*);
        __device__ void go(double *);
        __device__ void record(double *);
    private:
        __device__ void onestep();
        __device__ void getMean();
        __device__ void da(double,double*,double*,double*,double*);

        int tN_,numdt_,i_;
//        int N_;
        double h_,sigma_,ri_,omega_,phi_,t_;
        double *a1_,*a2_,*k_[4],*l_[4],*ta1_,*ta2_,*X_;
        double outMo_,outX_;
};

__device__
RNHQS_ZB::RNHQS_ZB()
{
}

__device__
void RNHQS_ZB::initial(int tN,double h,double sigma,double ri,double w,double phi,int numdt,double* X,double* a1,double* a2,double* ta1,double* ta2,double* k,double* l)
{
    tN_=tN;
    h_=h;
    sigma_=sigma;
    ri_=ri;
    omega_=w;
    phi_=phi;
    numdt_=numdt;
//    N_=200;
    i_=0;

    t_=0.0;

//    a1_=(double *)malloc(N_*sizeof(double));
//    a2_=(double *)malloc(N_*sizeof(double));
//    X_=(double *)malloc(N_*sizeof(double));
//    ta1_=(double *)malloc(N_*sizeof(double));
//    ta2_=(double *)malloc(N_*sizeof(double));
    a1_=a1;
    a2_=a2;
    X_=X;
    ta1_=ta1;
    ta2_=ta2;
    int i;
//    for (i=0;i<4;i++)
//    {
//        *(k_+i)=new double [N_];
//        *(l_+i)=new double [N_];
//    }

    for (i=0;i<4;i++)
    {
        *(k_+i)=k+N_*i;
        *(l_+i)=l+N_*i;
    }
    double w_d=6.0;
    for (i=0;i<N_;i++)
    {
        *(X_+i)=-N_/2.0+1.0/2.0+i;
        *(a1_+i)=exp(-pow(*(X_+i)/w_d,2.0))*cos(M_PI**(X_+i)/2.0);
        *(a2_+i)=exp(-pow(*(X_+i)/w_d,2.0))*sin(M_PI**(X_+i)/2.0);
    }
}

__device__ 
void RNHQS_ZB::go(double *_outMo)
{
    int ntime;

    for (ntime=0;ntime<numdt_;ntime++)
    {
        t_+=h_;
        RNHQS_ZB::onestep();
        if (ntime%1==0)
        {
            RNHQS_ZB::record(_outMo);
        }
    }
}

__device__ 
void RNHQS_ZB::onestep()
{
    int i,j;
    for (i=0;i<4;i++)
    {
        if (i==0)
        {
            RNHQS_ZB::da(t_,a1_,a2_,*(k_+i),*(l_+i));
        }
        else if (i==1 || i==2)
        {
            for (j=0;j<N_;j++)
            {
                *(ta1_+j)=*(a1_+j)+h_/2.0**(*(k_+i-1)+j);
                *(ta2_+j)=*(a2_+j)+h_/2.0**(*(l_+i-1)+j);
            }
            RNHQS_ZB::da(t_+h_/2.0,ta1_,ta2_,*(k_+i),*(l_+i));
        }
        else if (i==3)
        {
            for (j=0;j<N_;j++)
            {
                *(ta1_+j)=*(a1_+j)+h_**(*(k_+i-1)+j);
                *(ta2_+j)=*(a2_+j)+h_**(*(l_+i-1)+j);
            }
            RNHQS_ZB::da(t_+h_,ta1_,ta2_,*(k_+i),*(l_+i));
        }
    }
    for (j=0;j<N_;j++)
    {
        *(a1_+j)+=h_/6.0*(*(*k_+j)+2.0**(*(k_+1)+j)+2.0**(*(k_+2)+j)+*(*(k_+3)+j));
        *(a2_+j)+=h_/6.0*(*(*l_+j)+2.0**(*(l_+1)+j)+2.0**(*(l_+2)+j)+*(*(l_+3)+j));
    }
}

__device__ 
void RNHQS_ZB::da(double t,double *a1,double *a2,double *k,double *l)
{
    int i;
    double sigmai;

    sigmai=sigma_*ri_*sin(omega_*t+M_PI*phi_);

    for (i=1;i<N_-1;i++)
    {
        *(k+i)=-(*(a2+i+1)+*(a2+i-1))+pow(-1.0,i)*(sigma_**(a2+i)+sigmai**(a1+i));
        *(l+i)=(*(a1+i+1)+*(a1+i-1))+pow(-1.0,i+1)*(sigma_**(a1+i)-sigmai**(a2+i));
    }

    *(k+0)=-(*(a2+1)+*(a2+N_-1))+pow(-1.0,0)*(sigma_**(a2+0)+sigmai**(a1+0));
    *(l+0)=(*(a1+1)+*(a1+N_-1))+pow(-1.0,0+1)*(sigma_**(a1+0)-sigmai**(a2+0));

    *(k+N_-1)=-(*(a2+0)+*(a2+N_-2))+pow(-1.0,N_-1)*(sigma_**(a2+N_-1)+sigmai**(a1+N_-1));
    *(l+N_-1)=(*(a1+0)+*(a1+N_-2))+pow(-1.0,N_)*(sigma_**(a1+N_-1)-sigmai**(a2+N_-1));
}

__device__ 
void RNHQS_ZB::getMean()
{
    int i;
    outMo_=0.0;
    outX_=0.0;
    for (i=0;i<N_;i++)
    {
        outMo_+=pow(*(a1_+i),2.0)+pow(*(a2_+i),2.0);
        outX_+=*(X_+i)*(pow(*(a1_+i),2.0)+pow(*(a2_+i),2.0));
    }

//    if (wasExecuted_)
//   {
//        return;
//    }
//    else
//    {
//        double rest;
//        for (i=N_-10;i<N_-1;i++)
//        {
//            rest+=pow(*(a1_+i),2.0)+pow(*(a2_+i),2.0);
//        }

//        if (rest/outMo_>0.001)
//        {
 //           std::cout << tN_ << ": waveguide not long enough" << std::endl;
//        }
//        wasExecuted_ = true;
//    }
}

__device__ 
void RNHQS_ZB::record(double *_outMo)
{
    RNHQS_ZB::getMean();
    *(_outMo+i_)=outMo_;
    i_++;
}

__global__ void sRNHQS_ZB(int tN,double h,double sigma,double ri,double* omega,double phi,int numdt,double* _outMo,size_t pitch)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    RNHQS_ZB one;
//    __shared__ double a1[N_];
//    __shared__ double a2[N_];
//    __shared__ double X[N_];
//    __shared__ double ta1[N_];
//    __shared__ double ta2[N_];
//    __shared__ double k[4*N_];
//    __shared__ double l[4*N_];
    double a1[N_];
    double a2[N_];
    double X[N_];
    double ta1[N_];
    double ta2[N_];
    double k[4*N_];
    double l[4*N_];
    one.initial(tN,h,sigma,ri,*(omega+i),phi,numdt,X,a1,a2,ta1,ta2,k,l);
    one.go(_outMo+pitch/sizeof(double)*i);
}

int main (int argc, char *argv[])
{
    int tN,numdt,i,j;
    double h,phi,sigma,ri;
    double *omega;

    omega=new double [NTotal];
    for (i=0;i<NTotal;i++)
    {
        *(omega+i)=10.0/(NTotal-1.0)*i;
    }

//    size_t s1,s2,s3;
//    cudaDeviceSetLimit(cudaLimitStackSize,4*1024*sizeof(double));
    hipError_t cE;
    cE=hipDeviceSetLimit(hipLimitMallocHeapSize,16*1024*1024*sizeof(double));
    if (cE!=0) std::cout << cE << std::endl;
//    cudaDeviceSetLimit(cudaLimitPrintfFifoSize,1024*1024*sizeof(double));
//    cudaDeviceGetLimit(&s1,cudaLimitStackSize);
//    cudaDeviceGetLimit(&s2,cudaLimitPrintfFifoSize);
//    cudaDeviceGetLimit(&s3,cudaLimitMallocHeapSize);
//    std::cout << s1/sizeof(double)/1024/1024 << "M " << s2/sizeof(double)/1024/1024 << " " << s3/sizeof(double)/1024/1024 << std::endl;
//    std::cout << "Stack size per thread = " << s1/1024 << "Kb" << std::endl;
//    std::cout << "IO in total = " << s2/1024 << "Kb" << std::endl;
//    std::cout << "IO per thread = " << s2/1024/1024 << "Kb" << std::endl;
//    std::cout << "Heap size in total = " << s3/1024 << "Kb" << std::endl;
//    std::cout << "Heap size in total = " << s3/1024/1024 << "Mb" << std::endl;
//    std::cout << "Heap size per thread = " << s3/1024/1024 << "Kb" << std::endl;
//    std::cout << "Total double per thread = " << s3/sizeof(double)/1024  << std::endl;

    if (argc>1)
    {
        
        if (argc==7)
        {
            tN=(int)atof(argv[1]);
            h=(double)atof(argv[2]);
            sigma=(double)atof(argv[3]);
            ri=(double)atof(argv[4]);
            phi=(double)atof(argv[5]);
            numdt=(int)atof(argv[6]);

            double* _omega;
            cE=hipMalloc(&_omega,1024*sizeof(double));
            if (cE!=0) std::cout << cE << std::endl;
            cE=hipMemcpy(_omega,omega,1024*sizeof(double),hipMemcpyHostToDevice);
            if (cE!=0) std::cout << cE << std::endl;

            double *_outMo,*outMo;
            size_t pitch;
            cE=hipMallocPitch(&_outMo,&pitch,sizeof(double)*numdt,NTotal);
            if (cE!=0) std::cout << cE << std::endl;
            outMo=new double [numdt*NTotal];
            sRNHQS_ZB<<<Nblock,NThreadPerBlock>>>(tN,h,sigma,ri,_omega,phi,numdt,_outMo,pitch);
            cE=hipDeviceSynchronize();
            if (cE!=0) std::cout << cE << std::endl;
            std::cout << "CUDA computation complete. Start to copy data into host." << std::endl;
            cE=hipMemcpy2D(outMo,numdt*sizeof(double),_outMo,pitch,numdt*sizeof(double),NTotal,hipMemcpyDeviceToHost);
            if (cE!=0) std::cout << cE << std::endl;
            std::cout << "Copy finished. Start to write to file." << std::endl;

            char* filename[NTotal];
            std::ofstream out[NTotal];
            for (i=0;i<NTotal;i++)
            {
                filename[i]=new char [20];
                sprintf(filename[i],"ZB%d.txt",i+1);
                out[i].open(filename[i],std::ostream::out);
                for (j=0;j<numdt;j++)
                {
                    out[i] << *((double *)outMo+i*numdt+j) << std::endl;
                }
            }
        }
    }

    std::cout << "time used = " << clock()/CLOCKS_PER_SEC << std::endl;

    return 0;
}
